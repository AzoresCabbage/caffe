
#include <hip/hip_runtime.h>
//#include <vector>
//#include <algorithm>
//#include <cmath>
//
//#include "caffe/layers/conv_gru_layer.hpp"
//#include "caffe/util/math_functions.hpp"
//
//namespace caffe {
//
//template <typename Dtype>
//__device__ Dtype sigmoid(const Dtype x) {
//	return Dtype(1) / (Dtype(1) + exp(-x));
//}
//
//template <typename Dtype>
//__device__ Dtype d_sigmoid(const Dtype x) {
//	return x * (1 - x);
//}
//
//template <typename Dtype>
//__device__ Dtype relu(const Dtype x) {
//	return max(x, Dtype(0));
//}
//
//template <typename Dtype>
//__device__ Dtype d_relu(const Dtype x) {
//	return x > 0 ? 1 : 0;
//}
//
//template <typename Dtype>
//__device__ Dtype tanh(const Dtype x) {
//	return Dtype(2) * sigmoid(Dtype(2) * x) - Dtype(1);
//}
//
//template <typename Dtype>
//__device__ Dtype d_tanh(const Dtype x) {
//	return 1 - x * x;
//}
//
//template <typename Dtype>
//__device__ Dtype hard_sigmoid(const Dtype x) {
//	return max(min(0.2 * x + 0.5, Dtype(1)), Dtype(0));
//}
//
//template <typename Dtype>
//__device__ Dtype d_hard_sigmoid(const Dtype x) {
//	if (x >= 1 || x <= 0)
//		return 0;
//	else
//		return 0.2;
//}
//
//template <typename Dtype>
//__global__ void TanHForward(const int nthreads, Dtype* data) {
//	CUDA_KERNEL_LOOP(index, nthreads) {
//		data[index] = tanh(data[index]);
//	}
//}
//
//template <typename Dtype>
//__global__ void TanHBackward(const int nthreads, Dtype* diff, Dtype* data) {
//	CUDA_KERNEL_LOOP(index, nthreads) {
//		diff[index] = diff[index] * d_tanh(data[index]);
//	}
//}
//
////nthreads : N * H ; N is seq num
////H : feature_dims = Channel*Height*Width
////hidden_pre_gate: the data after hidden conv, 2 types (Ur, Uz) for C*H*W
////input_pre_gate : the data after input conv. 3 types (Wr, Wz, W) for C*H*W
//template <typename Dtype>
//__global__ void SigmoidForward(const int nthreads, const int H, Dtype* hidden_pre_gate,
//	Dtype* input_pre_gate, Dtype* h_t_1, Dtype* hidden_reset_) {
//	CUDA_KERNEL_LOOP(index, nthreads) {
//        const int n = index / H;
//		const int d = index % H;
//
//		// Rt = sigmoid(Wr*Xt + Ur*H[t-1])
//		input_pre_gate[n * 3 * H + d] += hidden_pre_gate[n * 2 * H + d];
//		input_pre_gate[n * 3 * H + d] = sigmoid(input_pre_gate[n * 3 * H + d]);
//
//		// Zt = sigmoid(Wz*Xt + Uz*H[t-1])
//        input_pre_gate[n * 3 * H + H + d] += hidden_pre_gate[n * 2 * H + H + d];
//        input_pre_gate[n * 3 * H + H + d] = sigmoid(input_pre_gate[n * 3 * H + H + d]);
//
//		// Rt .* H[t-1] for before Ht_candidate conv
//        hidden_reset_[index] = input_pre_gate[n * 3 * H + d] * h_t_1[index];
//	}
//}
//
//template <typename Dtype>
//__global__ void ActivationForward(const int nthreads, const int H, Dtype* hidden_rt_pre_gate,
//	Dtype* input_pre_gate, Dtype* h_t_1, Dtype* h_t) {
//	CUDA_KERNEL_LOOP(index, nthreads) {
//        const int n = index / H;
//		const int d = index % H;
//        input_pre_gate[n * 3 * H + 2 * H + d] += hidden_rt_pre_gate[index];
//		
//		// Yujie: Why use relu here ?
//        //input_pre_gate[n * 3 * H + 2 * H + d] = relu(input_pre_gate[n * 3 * H + 2 * H + d]);
//		input_pre_gate[n * 3 * H + 2 * H + d] = tanh(input_pre_gate[n * 3 * H + 2 * H + d]);
//
//        Dtype z_t = input_pre_gate[n * 3 * H + H + d];
//		h_t[index] = (1 - z_t) * h_t_1[index] + z_t * input_pre_gate[n * 3 * H + 2 * H + d];
//	}
//}
//
//template <typename Dtype>
//__global__ void ActivationBackward(const int nthreads, const int H,
//	const Dtype* gate, Dtype* pre_gate_diff, Dtype* hidden_reset_,
//	const Dtype* h_t_1, Dtype* dh_t_1, const Dtype* dh_t,
//    Dtype* hidden_rt_pre_gate_diff) {
//	CUDA_KERNEL_LOOP(index, nthreads) {
//        const int n = index / H;
//		const int d = index % H;
//		
//		dh_t_1[index] += dh_t[index] * (1 - gate[n * 3 * H + H + d]);
//
//		pre_gate_diff[n * 3 * H + 2 * H + d] = dh_t[index] * gate[n * 3 * H + H + d];
//		// Yujie : Why use relu here?
//		// pre_gate_diff[n * 3 * H + 2 * H + d] *= d_relu(gate[n * 3 * H + 2 * H + d]);
//		pre_gate_diff[n * 3 * H + 2 * H + d] *= d_tanh(gate[n * 3 * H + 2 * H + d]);
//
//		pre_gate_diff[n * 3 * H + H + d] = dh_t[index] * (gate[n * 3 * H + 2 * H + d] - h_t_1[index]);
//		pre_gate_diff[n * 3 * H + H + d] *= d_sigmoid(gate[n * 3 * H + H + d]);
//
//	    hidden_reset_[index] = gate[n * 3 * H + d] * h_t_1[index];
//
//        hidden_rt_pre_gate_diff[index] = pre_gate_diff[n * 3 * H + 2 * H + d];
//    }
//}
//
//template <typename Dtype>
//__global__ void SigmoidBackward(const int nthreads, const int H,
//	const Dtype* gate, Dtype* pre_gate_diff, const Dtype* h_t_1,
//    Dtype* dh_t_1, const Dtype* hidden_rt_diff, Dtype* hidden_pre_gate_diff) {
//	CUDA_KERNEL_LOOP(index, nthreads) {
//        const int n = index / H;
//		const int d = index % H;
//		
//        dh_t_1[index] += hidden_rt_diff[index] * gate[n * 3 * H + d];
//        pre_gate_diff[n * 3 * H + d] = hidden_rt_diff[index] * h_t_1[index];
//        pre_gate_diff[n * 3 * H + d] *= d_sigmoid(gate[n * 3 * H + d]);
//
//        hidden_pre_gate_diff[n * 2 * H + d] = pre_gate_diff[n * 3 * H + d];
//        hidden_pre_gate_diff[n * 2 * H + H + d] = pre_gate_diff[n * 3 * H + H + d];
//    }
//}
//
//template <typename Dtype>
//void ConvGRULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
//	const vector<Blob<Dtype>*>& top) {
//	Dtype* top_data = top[0]->mutable_gpu_data();
//
//	Dtype* input_pre_gate_data = input_pre_gate_.mutable_gpu_data();
//	Dtype* hidden_pre_gate_data = hidden_pre_gate_.mutable_gpu_data();
//	Dtype* hidden_rt_data = hidden_reset_.mutable_gpu_data();
//	Dtype* hidden_rt_pre_gate_data = hidden_rt_pre_gate_.mutable_gpu_data();
//	int feature_dims = H_ * spatial_dims;
//
//
//
//	// Compute input to gate forward propagation
//	conv_input_layer_->Forward(conv_input_bottom_vec_, conv_input_top_vec_);
//
//	// Initialize previous state
//	if (bottom.size() == 2)
//	{
//		h_0_.ShareData(*(bottom[1]));
//		h_0_.ShareDiff(*(bottom[1]));
//	}
//	else
//	{
//		// caffe_gpu_set(h_0_.count(0), Dtype(0.), h_0_.mutable_gpu_data());
//		// set H0 as W*X[t]
//		caffe_copy(h_0_.count(0), input_pre_gate_data + 2 * feature_dims, h_0_.mutable_gpu_data());
//		TanHForward<Dtype> << <CAFFE_GET_BLOCKS(N_ * feature_dims), CAFFE_CUDA_NUM_THREADS >> >(h_0_.count(0), h_0_.mutable_gpu_data());
//		CUDA_POST_KERNEL_CHECK;
//	}
//
//	// Compute recurrent forward propagation
//	for (int tt = 0; tt < T_; ++tt) {
//		int t = tt;
//		if (!forward_direction_) t = T_ - tt - 1;
//
//		Dtype* h_t = top_data + top[0]->count(1) * t;
//		Dtype* input_pre_gate_t = input_pre_gate_data + input_pre_gate_.count(1) * t;
//
//		Dtype* h_t_1 = t > 0 ? (h_t - top[0]->count(1)) : h_0_.mutable_gpu_data();
//
//		if (!forward_direction_){
//			h_t_1 = t < T_ - 1 ? (h_t + top[0]->count(1)) : h_0_.mutable_gpu_data();
//		}
//
//		// Hidden-to-hidden propagation
//		hidden_.data()->set_gpu_data(h_t_1);
//		conv_hidden_layer_->Forward(conv_hidden_bottom_vec_, conv_hidden_top_vec_);
//
//        SigmoidForward<Dtype><<<CAFFE_GET_BLOCKS(N_ * feature_dims), CAFFE_CUDA_NUM_THREADS>>>(N_ * feature_dims, feature_dims,
//	        hidden_pre_gate_data, input_pre_gate_t, h_t_1, hidden_rt_data);
//		CUDA_POST_KERNEL_CHECK;
//
//        conv_tmp_hidden_layer_->Forward(conv_tmp_hidden_bottom_vec_, conv_tmp_hidden_top_vec_);
//
//        ActivationForward<Dtype><<<CAFFE_GET_BLOCKS(N_ * feature_dims), CAFFE_CUDA_NUM_THREADS>>>(N_ * feature_dims, feature_dims,
//	        hidden_rt_pre_gate_data, input_pre_gate_t, h_t_1, h_t);
//		CUDA_POST_KERNEL_CHECK;
//		if (top.size() > 1)
//		{
//			caffe_gpu_memcpy(input_pre_gate_.data()->size(), input_pre_gate_data, top[1]->mutable_gpu_data());
//		}
//	}
//}
//
//template <typename Dtype>
//void ConvGRULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
//	const vector<bool>& propagate_down,
//	const vector<Blob<Dtype>*>& bottom) {
//	const Dtype* gate_data = input_pre_gate_.gpu_data();
//
//	Dtype* top_diff = top[0]->mutable_gpu_diff();
//	Dtype* pre_gate_diff = input_pre_gate_.mutable_gpu_diff();
//	Dtype* hidden_pre_gate_diff = hidden_pre_gate_.mutable_gpu_diff();
//	Dtype* hidden_rt_data = hidden_reset_.mutable_gpu_data();
//	Dtype* hidden_rt_pre_gate_diff = hidden_rt_pre_gate_.mutable_gpu_diff();
//	const Dtype* hidden_rt_diff = hidden_reset_.mutable_gpu_diff();
//	caffe_gpu_set(h_0_.count(0), Dtype(0.), h_0_.mutable_gpu_diff());
//
//	int feature_dims = H_ * spatial_dims;
//
//	for (int tt = T_ - 1; tt >= 0; --tt) {
//		int t = tt;
//		if (!forward_direction_) t = T_ - tt - 1;
//
//		Dtype* dh_t = top_diff + top[0]->count(1) * t;
//		Dtype* pre_gate_diff_t = pre_gate_diff + input_pre_gate_.count(1) * t;
//		const Dtype* gate_t = gate_data + input_pre_gate_.count(1) * t;
//
//		Dtype* dh_t_1 = t > 0 ? top_diff + top[0]->count(1) * (t - 1) : h_0_.mutable_gpu_diff();
//		Dtype* h_t_1 = t > 0 ? (top[0]->mutable_gpu_data() + top[0]->count(1) * (t - 1)) : h_0_.mutable_gpu_data();
//		if (!forward_direction_){
//			dh_t_1 = t < T_ - 1 ? top_diff + top[0]->count(1) * (t + 1) : h_0_.mutable_gpu_diff();
//			h_t_1 = t < T_ - 1 ? (top[0]->mutable_gpu_data() + top[0]->count(1) * (t + 1)) : h_0_.mutable_gpu_data();
//		}
//
//        ActivationBackward<Dtype><<<CAFFE_GET_BLOCKS(N_ * feature_dims), CAFFE_CUDA_NUM_THREADS>>>(
//            N_ * feature_dims, feature_dims, gate_t, pre_gate_diff_t, hidden_rt_data,
//            h_t_1, dh_t_1, dh_t, hidden_rt_pre_gate_diff);
//        CUDA_POST_KERNEL_CHECK;
//
//        conv_tmp_hidden_layer_->Backward(conv_tmp_hidden_top_vec_, vector<bool>{true}, conv_tmp_hidden_bottom_vec_);
//
//        SigmoidBackward<Dtype><<<CAFFE_GET_BLOCKS(N_ * feature_dims), CAFFE_CUDA_NUM_THREADS>>>
//        (N_ * feature_dims, feature_dims, gate_t, pre_gate_diff_t, h_t_1, dh_t_1, hidden_rt_diff, hidden_pre_gate_diff);
//        CUDA_POST_KERNEL_CHECK;
//
//		// Backprop errors to the previous time step
//		hidden_.data()->set_gpu_data(h_t_1);
//		conv_hidden_layer_->Backward(conv_hidden_top_vec_, vector<bool>{true}, conv_hidden_bottom_vec_);
//		const Dtype* hidden_diff_ = hidden_.gpu_diff();
//		caffe_gpu_add<Dtype>(N_ * feature_dims, dh_t_1, hidden_diff_, dh_t_1);
//	}
//	if (bottom.size() == 1)
//	{
//		TanHBackward<Dtype> << <CAFFE_GET_BLOCKS(N_ * feature_dims), CAFFE_CUDA_NUM_THREADS >> >(h_0_.count(0), h_0_.mutable_gpu_diff(), h_0_.mutable_gpu_data());
//		CUDA_POST_KERNEL_CHECK;
//		caffe_gpu_add<Dtype>(h_0_.count(0), h_0_.mutable_gpu_diff(), pre_gate_diff + 2 * feature_dims, pre_gate_diff + 2 * feature_dims);
//	}
//	// Gradient w.r.t. bottom data
//	conv_input_layer_->Backward(conv_input_top_vec_, vector<bool>{propagate_down[0]}, conv_input_bottom_vec_);
//
//}
//
//INSTANTIATE_LAYER_GPU_FUNCS(ConvGRULayer);
//
//}  // namespace caffe